#include "hip/hip_runtime.h"
/*
 ============================================================================

 Author      : Artyom Lyan
 Version     :
 Copyright   : Shareable, my bachelor degree work
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>

#include <hip/hip_vector_types.h>
#include <ctime>

#include <iostream>
#include <numeric>
#include <cstdlib>
#include <string>
#include <map>
#include <cmath>
#include <cstdio>

#include "image_statistic.h"

using namespace std;

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

#define PRINTER(name) printer(#name, (name))

void printer(char *name, double value) {
	//printf("name: %s\tvalue: %lf\n", name, value);
	cout << name << "|" << value << endl;
}

/**
 * function returns size of matrix defined higher in bytes
 */
int get_matrix_size_bytes(int cols, int rows) {
	return (cols * rows) * sizeof(int);
}

/**
 * this is function that returns image density at point
 * m1 is column number
 * m2 is position in row
 */
__device__ int f(int m1, int m2, int cols, int rows, int* pic) {
	if (m1 < rows && m2 < cols && m1 >= 0 && m2 >= 0) {
		return pic[m1 * rows + m2];
	} else {
		return -1;
	}
}

/**
 * this is indicator function
 * m1,m2 are points
 * d_m1 and d_m2 are distances for 2 points
 * i and j are densities for m1,m2 and m1+d_m1, m2_d_m2 points respectively
 */
__device__ int q_ij(int m1, int m2, int d_m1, int d_m2, int i, int j, int cols,
		int rows, int* pic) {
	int result = 0;
	if (f(m1, m2, cols, rows, pic) == i
			&& f(m1 + d_m1, m2 + d_m2, cols, rows, pic) == j) {
		result = 1;
	}
	return result;
}

/**
 * this function returns non-normed values of adjacency matrix
 * here i and j are intensity levels
 * this function will be called once for single row and string
 * M2 is length of string
 * function will be launched N times, N - number of rows
 * c_values stores source image in format of vector
 */
__global__ void c_dm1_dm2(int i, int j, int cols, int rows, int dm1, int dm2,
		int* c_values, int* res) {
	int index = threadIdx.x; // in current implementation that's enough
	res[index] = 0;

	for (int m1 = 0; m1 < cols; m1++) {
		res[index] += q_ij(index, m1, dm1, dm2, i, j, cols, rows, c_values);
	}
}

/**
 * result will be stored in first vector
 */
__global__ void sum_vector(double* v1, double* v2, int len, int stride) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	for (int i = index * stride; i < (index + 1) * stride; i++) {
		v1[i] += v2[i];
	}
}

__global__ void devide_vector(double* v1, double devide_by, int stride) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;

	for (int i = index * stride; i < (index + 1) * stride; i++) {
		v1[i] /= devide_by;
	}
}

/**
 * in this program i handle each string of picture in single thread
 */

double* calc_adjacency_matrix(int dm1, int dm2, int* values, int cols, int rows,
		int max_i, int max_j) {

	int m_size = cols * rows * sizeof(int);

	int* c_values;

	CUDA_CHECK_RETURN(hipMalloc((void** )&c_values, m_size));
	CUDA_CHECK_RETURN(
			hipMemcpy(c_values, values, m_size, hipMemcpyHostToDevice));

	int* c_res;
	CUDA_CHECK_RETURN(hipMalloc((void** )&c_res, rows * sizeof(int)));

	int* res = new int[rows];

	int** res_mat = new int*[max_i];

	int total_pairs = 0;

	for (int i = 0; i < max_i; i++) {
		res_mat[i] = new int[max_j];
		for (int j = 0; j < max_j; j++) {
			c_dm1_dm2<<<1, rows>>>(i, j, cols, rows, dm1, dm2, c_values, c_res);
			CUDA_CHECK_RETURN(
					hipMemcpy(res, c_res, rows * sizeof(int),
							hipMemcpyDeviceToHost));

			res_mat[i][j] = std::accumulate(res, res + rows, 0);

			total_pairs += res_mat[i][j];
		}
	}

	double* normalized_res_mat = new double[max_i * max_j];
	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			normalized_res_mat[i * max_i + j] = (double) res_mat[i][j]
					/ total_pairs;
		}
	}

	CUDA_CHECK_RETURN(hipFree(c_values));
	CUDA_CHECK_RETURN(hipFree(c_res));

	delete[] res;
	for (int i = 0; i < max_i; i++) {
		delete[] res_mat[i];
	}
	delete[] res_mat;

	return normalized_res_mat;
}

double* calc_symmetric_adjacency_matrix(int* pic, int cols, int rows, int dm1,
		int dm2, int max_i, int max_j) {

	cout << "calc_symmetric_adjacency_matrix started" << endl;

	int size = max_i * max_j;

	double* adj_matr_1 = calc_adjacency_matrix(dm1, dm2, pic, cols, rows, max_i,
			max_j);
	cout << "1" << endl;
	double* adj_matr_2 = calc_adjacency_matrix(-dm1, dm2, pic, cols, rows,
			max_i, max_j);
	cout << "2" << endl;
	double* adj_matr_3 = calc_adjacency_matrix(dm1, -dm2, pic, cols, rows,
			max_i, max_j);
	cout << "3" << endl;
	double* adj_matr_4 = calc_adjacency_matrix(-dm1, -dm2, pic, cols, rows,
			max_i, max_j);
	cout << "4" << endl;

	double *c_adj_matr1, *c_adj_matr2, *c_adj_matr3, *c_adj_matr4;

	CUDA_CHECK_RETURN(hipMalloc((void** )&c_adj_matr1, size * sizeof(double)));
	CUDA_CHECK_RETURN(hipMalloc((void** )&c_adj_matr2, size * sizeof(double)));
	CUDA_CHECK_RETURN(hipMalloc((void** )&c_adj_matr3, size * sizeof(double)));
	CUDA_CHECK_RETURN(hipMalloc((void** )&c_adj_matr4, size * sizeof(double)));

	CUDA_CHECK_RETURN(
			hipMemcpy(c_adj_matr1, adj_matr_1, size * sizeof(double),
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(c_adj_matr2, adj_matr_2, size * sizeof(double),
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(c_adj_matr3, adj_matr_3, size * sizeof(double),
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(c_adj_matr4, adj_matr_4, size * sizeof(double),
					hipMemcpyHostToDevice));

	cout << "summing" << endl;
	sum_vector<<<1, max_i>>>(c_adj_matr1, c_adj_matr2, size, max_i);
	sum_vector<<<1, max_i>>>(c_adj_matr1, c_adj_matr3, size, max_i);
	sum_vector<<<1, max_i>>>(c_adj_matr1, c_adj_matr4, size, max_i);

	cout << "deviding" << endl;
	devide_vector<<<1, max_i>>>(c_adj_matr1, 4.0, max_i);

	CUDA_CHECK_RETURN(
			hipMemcpy(adj_matr_1, c_adj_matr1, size * sizeof(double),
					hipMemcpyDeviceToHost));

	delete[] adj_matr_2;
	delete[] adj_matr_3;
	delete[] adj_matr_4;

	CUDA_CHECK_RETURN(hipFree(c_adj_matr1));
	CUDA_CHECK_RETURN(hipFree(c_adj_matr2));
	CUDA_CHECK_RETURN(hipFree(c_adj_matr3));
	CUDA_CHECK_RETURN(hipFree(c_adj_matr4));

	cout << "finished" << endl;

	return adj_matr_1;
}

double first_angle_moment(double* matr, int max_i, int max_j, int i) {
	double result = 0;
	for (int j = 0; j < max_j; j++) {
		result += matr[i * max_i + j];
	}
	return result;
}

double mi(double* matr, int max_i, int max_j, int i) {
	double result = 0;
	for (int j = 0; j < max_j; j++) {
		result += j * matr[i * max_i + j] - i;
	}
	return result;
}

double mj(double* matr, int max_i, int max_j, int i) {
	double result = 0;
	for (int j = 0; j < max_j; j++) {
		result += i * matr[i * max_i + j] - j;
	}
	return result;
}

double first_main_moment(double* matr, int rows, int cols, int max_i,
		int max_j) {
	double result = 0;
	for (int i = 0; i < max_i; i++) {
		result += first_angle_moment(matr, max_i, max_j, i);
	}
	return result;
}

/**
 * warn, that result is powered by 2
 */
double second_angle_moment(double *matr, int max_i, int max_j) {

	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += pow(matr[i * max_i + j], 2.0);
		}
	}
	return result;
}

double contrast(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += abs(i - j) * matr[i * max_i + j];
		}
	}

	return result;
}

double intertion(double* matr, int max_i, int max_j) {

	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += pow(i - j, 2.0) * matr[i * max_i + j];
		}
	}

	return result;
}

double correlation(double* matr, int max_i, int max_j, double mx) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += (i - mx) * (j - mx) * matr[i * max_i + j];
		}
	}

	return result;
}

double blackout(double* matr, int max_i, int max_j, double mx) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += pow(i + j - 2 * mx, 3.0) * matr[i * max_i + j];
		}
	}

	return result;
}

double entropy(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += log(matr[i * max_i + j]) * matr[i * max_i + j];
		}
	}

	return result;
}

double backward_deviation(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += pow(1 + abs(i - j), -1.0) * matr[i * max_i + j];
		}
	}

	return result;
}

double backward_moment(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += pow(1 + pow(i - j, 2.0), -1.0) * matr[i * max_i + j];
		}
	}

	return result;
}

double diagonal_moment(double* matr, int max_i, int max_j, double mx) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += abs(i - j) * (i + j - 2 * mx) * matr[i * max_i + j];
		}
	}

	return result;
}

double summary_average(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += mi(matr, max_i, max_j, i)
					* first_angle_moment(matr, max_i, max_j, i);
		}
	}
	return result;
}

double summary_entropy(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += log(first_angle_moment(matr, max_i, max_j, i))
					* first_angle_moment(matr, max_i, max_j, i);
		}
	}
	return result;
}

double summary_correlation(double* matr, int max_i, int max_j) {
	double result = 0;

	for (int i = 0; i < max_i; i++) {
		for (int j = 0; j < max_j; j++) {
			result += mi(matr, max_i, max_j, i) * mj(matr, max_i, max_j, j);
		}
	}
	return result;
}

void test() {
	double* v1 = new double[4];
	double* v2 = new double[4];

	for (int i = 0; i < 4; i++) {
		v1[i] = i;
		v2[i] = i;
	}

	print_vector(v1, 4);
	print_vector(v2, 4);

	double *c_v1, *c_v2;

	hipMalloc((void**) &c_v1, 4 * sizeof(double));
	hipMalloc((void**) &c_v2, 4 * sizeof(double));

	hipMemcpy(c_v1, v1, 4 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(c_v2, v2, 4 * sizeof(double), hipMemcpyHostToDevice);

	sum_vector<<<1, 2>>>(c_v1, c_v2, 4, 2);

	hipMemcpy(v1, c_v1, 4 * sizeof(double), hipMemcpyDeviceToHost);
	print_vector(v1, 4);

	hipFree(c_v1);
	hipFree(c_v2);
	delete[] v1;
	delete[] v2;
}

double* calc_signs(double* adj_matr, int cols, int rows, int dm1, int dm2,
		int max_i, int max_j) {

	cout << "started calculation" << endl;

	int signs_num = 14;

	double *result = new double[signs_num];

	result[0] = signs_num;
	double fmm = result[1] = first_main_moment(adj_matr, rows, cols, max_i,
			max_j);
	result[2] = second_angle_moment(adj_matr, max_i, max_j);
	result[3] = contrast(adj_matr, max_i, max_j);
	result[4] = intertion(adj_matr, max_i, max_j);
	result[5] = correlation(adj_matr, max_i, max_j, fmm);
	result[6] = blackout(adj_matr, max_i, max_j, fmm);
	result[7] = entropy(adj_matr, max_i, max_j);
	result[8] = backward_deviation(adj_matr, max_i, max_j);
	result[9] = backward_moment(adj_matr, max_i, max_j);
	result[10] = diagonal_moment(adj_matr, max_i, max_j, fmm);
	result[11] = summary_average(adj_matr, max_i, max_j);
	result[12] = summary_correlation(adj_matr, max_i, max_j);
	result[13] = summary_entropy(adj_matr, max_i, max_j);

	return result;
}

void print_vector(double* v, int cols, int rows) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			cout << v[i * rows + j] << "|";
		}
		cout << endl;
	}
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

/**
 * this method prepares image (currently matrix)
 * to be proceed on cuda
 * unfortunately i didn't recognize how to handle
 * matrix in form of 2 dim array
 * so i use vector
 */
int* prepare_matrix(int cols, int rows) {
	int* values = new int[rows * cols];
	for (int i = 0; i < cols; i++) {
		for (int j = 0; j < rows; j++) {
			values[i * cols + j] = rand() % 10;
		}
	}
	return values;
}

void print_vector(int* v, int len) {
	for (int i = 0; i < len; i++) {
		cout << v[i] << "|";
	}
	cout << endl;
}
void print_vector(double* v, int len) {
	for (int i = 0; i < len; i++) {
		cout << v[i] << "|";
	}
	cout << endl;
}

void print_matrix(int** m, int w, int h) {
	for (int i = 0; i < h; i++) {
		print_vector(m[i], w);
	}
}

void print_matrix(double** m, int w, int h) {
	for (int i = 0; i < h; i++) {
		print_vector(m[i], w);
	}
}

void print_matrix(matrix* matr) {
	for (int i = 0; i < matr->cols; i++) {
		for (int j = 0; j < matr->rows; j++) {
			cout << matr->values[i * matr->cols + j] << ":";
		}
		cout << endl;
	}
}
